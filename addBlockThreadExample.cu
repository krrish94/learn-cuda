
#include <hip/hip_runtime.h>
#include <iostream>


__global__ void add(int *a, int *b, int *c) {
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	c[index] = a[index] + b[index];
}


#define N (2048 * 2048)
#define THREADS_PER_BLOCK 512


int main(void) {

	// host copies of a, b, c
	int *a, *b, *c;
	// device copies of a, b, c
	int *d_a, *d_b, *d_c;

	int size = N * sizeof(int);

	// Alloc space and setup
	a = (int *) malloc(size);
	b = (int *) malloc(size);
	c = (int *) malloc(size);

	for(int i = 0; i < N; ++i) {
		a[i] = 2;
		b[i] = 7;
	}

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	// Copy inputs to device
	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	// Launch add() kernel on GPU
	add<<<N/THREADS_PER_BLOCK,THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

	// Copy result back to host
	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	// Cleanup
	free(a);
	free(b);
	free(c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;

}
